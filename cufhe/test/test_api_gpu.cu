/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include <include/cufhe_gpu.cuh>
#include <include/cufhe.h>
using namespace cufhe;

#include <iostream>
#include <vector>
using namespace std;

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void OrCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) > 0;
}

void AndCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = in0.message_ * in1.message_;
}

void XorCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) & 0x1;
}

int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
  uint32_t kNumTests = kNumSMs * 32;// * 8;
  uint32_t kNumLevels = 4;

  SetSeed(); // set random seed

  PriKey pri_key; // private key
  PubKey pub_key; // public key
  Ptxt* pt = new Ptxt[2 * kNumTests];
  Ctxt* ct = new Ctxt[2 * kNumTests];
  Synchronize();
  bool correct;

  cout<< "------ Key Generation ------" <<endl;
  ReadPubKeyFromFile(pub_key, "pub.key");
  ReadPriKeyFromFile(pri_key, "pri.key");
  //KeyGen(pub_key, pri_key);
  // Alternatively ...
  // PriKeyGen(pri_key);
  // PubKeyGen(pub_key, pri_key);

  // cout<< "------ Test Encryption/Decryption ------" <<endl;
  // cout<< "Number of tests:\t" << kNumTests <<endl;
  // correct = true;
  // for (int i = 0; i < kNumTests; i ++) {
  //   pt[i].message_ = rand() % Ptxt::kPtxtSpace;
  //   Encrypt(ct[i], pt[i], pri_key);
  //   Decrypt(pt[kNumTests + i], ct[i], pri_key);
  //   if (pt[kNumTests + i].message_ != pt[i].message_) {
  //     correct = false;
  //     break;
  //   }
  // }
  // if (correct)
  //   cout<< "PASS" <<endl;
  // else
  //   cout<< "FAIL" <<endl;

  // cout<< "------ Initilizating Data on GPU(s) ------" <<endl;
  Initialize(pub_key); // essential for GPU computing

  vector<Ctxt> ctxt(10);
  // Ptxt p;
  // p.message_ = 0;
  // for (int i = 0; i < 10; i++) {
  //   Encrypt(ctxt[i], p, pri_key);
  // }

  //WriteCtxtToFile(ctxt, "test.ctxt");

  //ReadCtxtFromFile(ctxt, "test.ctxt");
  WriteCtxtToFile(ctxt, "test1.ctxt");

  // cout<< "------ Test NAND Gate ------" <<endl;
  // cout<< "Number of tests:\t" << kNumTests <<endl;
  // // Create CUDA streams for parallel gates.
  // Stream* st = new Stream[kNumSMs];
  // for (int i = 0; i < kNumSMs; i ++)
  //   st[i].Create();

  // correct = true;
  // for (int i = 0; i < 2 * kNumTests; i ++) {
  //   pt[i] = rand() % Ptxt::kPtxtSpace;
  //   Encrypt(ct[i], pt[i], pri_key);
  // }
  // Synchronize();

  // float et;
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  // // Here, pass streams to gates for parallel gates.
  // for (int i = 0; i < kNumTests; i ++)
  //   Nand(ct[i], ct[i], ct[i + kNumTests], st[i % kNumSMs]);
  // for (int i = 0; i < kNumTests; i ++)
  //   Or(ct[i], ct[i], ct[i + kNumTests], st[i % kNumSMs]);
  // for (int i = 0; i < kNumTests; i ++)
  //   And(ct[i], ct[i], ct[i + kNumTests], st[i % kNumSMs]);
  // for (int i = 0; i < kNumTests; i ++)
  //   Xor(ct[i], ct[i], ct[i + kNumTests], st[i % kNumSMs]);
  // Synchronize();

  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // hipEventElapsedTime(&et, start, stop);
  // cout<< et / kNumTests / kNumLevels << " ms / gate" <<endl;
  // hipEventDestroy(start);
  // hipEventDestroy(stop);

  // int cnt_failures = 0;
  // for (int i = 0; i < kNumTests; i ++) {
  //   NandCheck(pt[i], pt[i], pt[i + kNumTests]);
  //   OrCheck(pt[i], pt[i], pt[i + kNumTests]);
  //   AndCheck(pt[i], pt[i], pt[i + kNumTests]);
  //   XorCheck(pt[i], pt[i], pt[i + kNumTests]);
  //   Decrypt(pt[i + kNumTests], ct[i], pri_key);
  //   if (pt[i + kNumTests].message_ != pt[i].message_) {
  //     correct = false;
  //     cnt_failures += 1;
  //     //std::cout<< "Fail at iteration: " << i <<std::endl;
  //   }
  // }
  // if (correct)
  //   cout<< "PASS" <<endl;
  // else
  //   cout<< "FAIL:\t" << cnt_failures << "/" << kNumTests <<endl;
  // for (int i = 0; i < kNumSMs; i ++)
  //   st[i].Destroy();
  // delete [] st;

  
  // delete [] ct;
  // delete [] pt;
  cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
  CleanUp(); // essential to clean and deallocate data
  return 0;
}
